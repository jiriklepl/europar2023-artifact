#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <cstring>

#define CUCH(status)  do { hipError_t err = status; if (err != hipSuccess) std::cerr << __FILE__ ":" << __LINE__ << ": error: " << hipGetErrorString(err) << "\n\t" #status << std::endl, exit(err); } while (false)

using num_t = float;

static constexpr std::size_t I_BLOCK_SIZE = 1024;
static constexpr std::size_t K_BLOCK_SIZE = 8;

enum layout { ROW_MAJOR, COL_MAJOR };

template<class Scalar, class Step, layout Layout>
struct matrix {
	constexpr matrix(Scalar *data, Step row_count, Step col_count) noexcept
		: data(data), step((Layout == ROW_MAJOR) ? row_count : col_count) {
	}

	template<class Major, class Minor>
	constexpr Scalar &operator() (Major major, Minor minor) const noexcept {
		if constexpr (Layout == ROW_MAJOR)
			return data[major * step + minor];
		else
			return data[major * step + minor];
	}

	Scalar *data;
	Step step;
};

template<auto Layout, class Scalar, class Step>
constexpr auto make_matrix(Scalar *data, Step row_count, Step col_count) noexcept {
	return matrix<Scalar, Step, Layout>(data, row_count, col_count);
}

template<std::size_t ISize, std::size_t JSize, std::size_t KSize, class A, class B, class C>
__global__ void kernel_matmul(A a, B b, C c) {
	extern __shared__ num_t shm_c[];
	auto d = make_matrix<ROW_MAJOR>(shm_c,  I_BLOCK_SIZE, K_BLOCK_SIZE);

	auto I = blockIdx.x * I_BLOCK_SIZE;
	auto K = blockIdx.y * K_BLOCK_SIZE;
	auto i = threadIdx.x;

	for(size_t k = 0; k < K_BLOCK_SIZE; k++) {
		d(k, i) = 0;
	}

	for(size_t j = 0; j < JSize; j++) {
		for(size_t k = 0; k < K_BLOCK_SIZE; k++) {
			d(k, i) += a(j, I + i) * b(K + k, j);
		}
	}

	for(size_t k = 0; k < K_BLOCK_SIZE; k++) {
		c(K + k, I + i) = d(k, i);
	}
}

template<std::size_t ISize, std::size_t JSize, std::size_t KSize, class A, class B, class C>
void matmul_cuda(A a, B b, C c) {
	kernel_matmul<ISize, JSize, KSize><<<{ISize/I_BLOCK_SIZE, KSize/K_BLOCK_SIZE}, I_BLOCK_SIZE, I_BLOCK_SIZE * K_BLOCK_SIZE * sizeof(float)>>>(a, b, c);
	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());
}

using namespace std::literals::chrono_literals;

int main(int argc, char **argv) {
	if(argc != 2) {
		std::cerr << "Usage" << std::endl;
		std::abort();
	}

	constexpr std::size_t ISize = 8192;
	constexpr std::size_t JSize = 8192;
	constexpr std::size_t KSize = 8192;

	std::size_t a_cnt = ISize * JSize;
	std::size_t b_cnt = JSize * KSize;
	std::size_t c_cnt = ISize * KSize;

	std::size_t a_sz = a_cnt * sizeof(num_t);
	std::size_t b_sz = b_cnt * sizeof(num_t);
	std::size_t c_sz = c_cnt * sizeof(num_t);

	num_t *data;
	CUCH(hipMallocManaged(&data, a_sz + b_sz + c_sz));

	std::FILE *file = std::fopen(argv[1], "r");
	if(std::fread(data, 1, a_sz + b_sz, file) != a_sz + b_sz) {
		std::cerr << "Input error" << std::endl;
		std::abort();
	}
	std::fclose(file);

	auto a = make_matrix<ROW_MAJOR>(
		data,
		std::integral_constant<std::size_t, ISize>(),
		std::integral_constant<std::size_t, JSize>());
	auto b = make_matrix<ROW_MAJOR>(
		data + a_cnt,
		std::integral_constant<std::size_t, JSize>(),
		std::integral_constant<std::size_t, KSize>());
	auto c = make_matrix<ROW_MAJOR>(
		data + a_cnt + b_cnt, 
		std::integral_constant<std::size_t, ISize>(),
		std::integral_constant<std::size_t, KSize>());

	matmul_cuda<ISize, JSize, KSize>(a, b, c);

	auto t0 = std::chrono::steady_clock::now();
	matmul_cuda<ISize, JSize, KSize>(a, b, c);
	auto t1 = std::chrono::steady_clock::now();
	std::fprintf(stderr, "%lu.%03u ms\n", (unsigned long) ((t1 - t0) / 1ms), (unsigned) ((t1 - t0) / 1us % 1000));

	std::fwrite(data + a_cnt + b_cnt, 1, c_sz, stdout);

	CUCH(hipFree(data));

	return 0;
}
