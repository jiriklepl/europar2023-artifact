#include "hip/hip_runtime.h"
#define CUDA
#include "noarrmain.hpp"

template<class T, class C>
__global__ void kernel_bzero(T trav, C c) {
	trav.for_each([&](auto state) {
		c[state] = 0;
	});
}

template<class T, class A, class B, class C>
__global__ void kernel_matmul(T trav, A a, B b, C c) {
	trav.for_each([&](auto state) {
		c[state] += a[state] * b[state];
	});
}

template<class A, class B, class C>
void matmul(A ta, B tb, C tc, char *pa, char *pb, char *pc) {
	auto a = noarr::make_bag(ta, pa);
	auto b = noarr::make_bag(tb, pb);
	auto c = noarr::make_bag(tc, pc);

	static constexpr auto I_BLOCK_SIZE = 32;
	static constexpr auto K_BLOCK_SIZE = 32;

	auto into_blocks = noarr::into_blocks_dynamic<'i', 'I', 'i', 'r'>(I_BLOCK_SIZE) ^ noarr::into_blocks_dynamic<'k', 'K', 'k', 's'>(K_BLOCK_SIZE);

	{
		auto trav = noarr::cuda_threads<'I', 'i', 'K', 'k'>(
			noarr::traverser(c).order(into_blocks)
			);

		kernel_bzero<<<trav.grid_dim(), trav.block_dim()>>>(trav.inner(), c);
		CUCH(hipGetLastError());
	}

	{
		auto trav = noarr::cuda_threads<'I', 'i', 'K', 'k'>(
			noarr::traverser(a, b, c).order(noarr::hoist<'i'>() ^ noarr::hoist<'k'>())
				.order(into_blocks)
			);

		kernel_matmul<<<trav.grid_dim(), trav.block_dim()>>>(trav.inner(), a, b, c);
		CUCH(hipGetLastError());
	}

	CUCH(hipDeviceSynchronize());
}
