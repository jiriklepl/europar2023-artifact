#include "hip/hip_runtime.h"
#define CUDA
#include "plainmain.hpp"

#ifndef BLOCK_SIZE
#error define appropriate BLOCK_SIZE
#endif

template<class ISize, class JSize, class KSize>
__global__ void kernel_matmul(ISize i_size, JSize j_size, KSize k_size, num_t* glm_a, num_t* glm_b, num_t* glm_c) {
	num_t result = 0;

	auto i = blockIdx.x * blockDim.x + threadIdx.x;
	auto k = blockIdx.y * blockDim.y + threadIdx.y;

	for (std::size_t j = 0; j < j_size; j++) {
		num_t local_a = glm_a[j*i_size + i];
		num_t local_b = glm_b[k*j_size + j];
		result += local_a * local_b;
	}

	glm_c[k*i_size + i] = result;
}

template<class ISize, class JSize, class KSize>
void matmul(ISize i_size, JSize j_size, KSize k_size, num_t* cu_a, num_t* cu_b, num_t* cu_c) {
	auto i_block_dim = uint((i_size - 1) / BLOCK_SIZE + 1);
	auto k_block_dim = uint((k_size - 1) / BLOCK_SIZE + 1);

	kernel_matmul<<<{i_block_dim, k_block_dim}, {(uint)BLOCK_SIZE, (uint)BLOCK_SIZE}>>>(i_size, j_size, k_size, cu_a, cu_b, cu_c);

	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());
}
