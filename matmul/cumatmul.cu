#include "hip/hip_runtime.h"
#define CUDA
#include "matmulmain.hpp"

template<typename T, typename TA, typename TB, typename TC, typename TD>
__global__ void kernel_matmul(T trav, TA ta, TB tb, TC tc, TD td, void *pa, void *pb, void *pc) {
	extern __shared__ char pd[];

	trav.template for_dims<'k'>([&](auto inner) {
		td | noarr::get_at(pd, inner.state()) = 0;
	});

	trav.template for_dims<'j', 'k'>([&](auto inner) {
		auto ijk = inner.state();
		num_t a_elem = ta | noarr::get_at(pa, ijk);
		num_t b_elem = tb | noarr::get_at(pb, ijk);
		td | noarr::get_at(pd, ijk) += a_elem * b_elem;
	});

	trav.template for_dims<'k'>([&](auto inner) {
		auto ik = inner.state();
		num_t c_elem = td | noarr::get_at(pd, ik);
		tc | noarr::get_at(pc, ik) = c_elem;
	});
}

template<typename A, typename B, typename C>
void matmul(A orig_ta, B orig_tb, C orig_tc, char *pa, char *pb, char *pc) {
	auto i_blocks = noarr::into_blocks<'i', /*'r',*/ 'I', 'i'>(noarr::lit<1024>);
	auto k_blocks = noarr::into_blocks<'k', /*'s',*/ 'K', 'k'>(noarr::lit<8>);

	auto ta = orig_ta ^ i_blocks;
	auto tb = orig_tb ^ k_blocks;
	auto tc = orig_tc ^ i_blocks ^ k_blocks;

	noarr::traverser(ta, tb, tc).template for_dims</*'s', 'r'*/>([=](auto trav) {
		auto td = noarr::scalar<num_t>()
			^ noarr::sized_vector<'i'>(tc | noarr::get_length<'i'>(trav.state()))
			^ noarr::sized_vector<'k'>(tc | noarr::get_length<'k'>(trav.state()));

		auto cutrav = noarr::cuda_threads<'I', 'i', 'K', '1'>(trav.order(noarr::bcast<'1'>(1)));
		kernel_matmul<<<cutrav.grid_dim(), cutrav.block_dim(), td | noarr::get_size()>>>(cutrav.inner(), ta, tb, tc, td, pa, pb, pc);

		CUCH(hipGetLastError());
	});

	CUCH(hipDeviceSynchronize());
}
