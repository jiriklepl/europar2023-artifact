#include "hip/hip_runtime.h"
#define CUDA
#include "noarrmain.hpp"

#include <noarr/structures/interop/bag.hpp>

#ifndef BLOCK_SIZE
#error define appropriate BLOCK_SIZE
#endif

template<class T, class A, class B, class C>
__global__ void matmul(T trav, A a, B b, C c) {
	trav.template for_dims<'r', 's'>([=](auto trav) {
		num_t result = 0;

		trav.template for_dims<'k'>([=, &result](auto inner) {
			auto state = inner.state();
			result += a[state] * b[state];
		});

		c[trav.state()] = result;
	});
}

template<class A, class B, class C>
void run_matmul(A ta, B tb, C tc, num_t *pa, num_t *pb, num_t *pc) {
	auto a = noarr::make_bag(ta, pa);
	auto b = noarr::make_bag(tb, pb);
	auto c = noarr::make_bag(tc, pc);

#ifdef DYNAMIC_BLOCKS
	auto into_blocks = noarr::into_blocks_dynamic<'i', 'I', 'i', 'r'>(noarr::lit<BLOCK_SIZE>) ^ noarr::into_blocks_dynamic<'j', 'J', 'j', 's'>(noarr::lit<BLOCK_SIZE>);
#else
	auto into_blocks = noarr::into_blocks<'i', 'I', 'i'>(noarr::lit<BLOCK_SIZE>) ^ noarr::into_blocks<'j', 'J', 'j'>(noarr::lit<BLOCK_SIZE>) ^ noarr::bcast<'r', 's'>(noarr::lit<1>, noarr::lit<1>);
#endif

	auto cutrav = noarr::cuda_threads<'I', 'i', 'J', 'j'>(noarr::traverser(a, b, c).order(into_blocks));

	cutrav.simple_run(matmul, 0, a, b, c);
	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());
}
