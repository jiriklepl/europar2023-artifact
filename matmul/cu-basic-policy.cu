#include "hip/hip_runtime.h"
#define CUDA
#include "policymain.hpp"

#ifndef BLOCK_SIZE
#error define appropriate BLOCK_SIZE
#endif

template<class ISize, class JSize, class KSize, class A, class B, class C>
__global__ void kernel_matmul(ISize i_size, JSize j_size, KSize k_size, A a, B b, C c) {
	num_t result = 0;

	auto i = blockIdx.x * blockDim.x + threadIdx.x;
	auto k = blockIdx.y * blockDim.y + threadIdx.y;

	for (std::size_t j = 0; j < j_size; j++) {
		result += a(j, i) * b(k, j);
	}

	c(k, i) = result;
}

template<class ISize, class JSize, class KSize, class A, class B, class C>
void matmul(ISize i_size, JSize j_size, KSize k_size, A a, B b, C c) {

	kernel_matmul<<<{(uint)(i_size/BLOCK_SIZE), (uint)(k_size/BLOCK_SIZE)}, {(uint)BLOCK_SIZE, (uint)BLOCK_SIZE}>>>(i_size, j_size, k_size, a, b, c);

	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());
}
