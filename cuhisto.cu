#include "hip/hip_runtime.h"
#include "cuhistomain.hpp"

template<typename InTrav, typename InStruct, typename ShmStruct, typename OutStruct>
__global__ void kernel_histo(InTrav in_trav, InStruct in_struct, ShmStruct shm_struct, OutStruct out_struct, void *in_ptr, void *out_ptr) {
	extern __shared__ char shm_ptr[];

	// A private copy will usually be shared by multiple threads (whenever NUM_COPIES < blockDim.x).
	// For some actions, we would like each memory location to be assigned to only one thread.
	// Let us split each copy further into "subsets", where each subset is owned by exactly one thread.
	std::size_t my_copy_idx = shm_struct.current_stripe_index();
	auto subset = noarr::cuda_step(shm_struct.current_stripe_cg());

	// Zero out shared memory. In this particular case, the access pattern happens
	// to be the same as with the `for(i = threadIdx; i < ...; i += blockDim)` idiom.
	noarr::traverser(shm_struct).order(subset).for_each([=](auto state) {
		shm_struct | noarr::get_at(shm_ptr, state) = 0;
	});

	__syncthreads();

	// Count the elements into the histogram copies in shared memory.
	in_trav.for_each([=](auto state) {
		auto value = in_struct | noarr::get_at(in_ptr, state);
		auto &bin = shm_struct | noarr::get_at<'v'>(shm_ptr, value);
		atomicAdd(&bin, 1);
	});

	__syncthreads();

	// Reduce the bins in shared memory into global memory.
	noarr::traverser(out_struct).order(noarr::cuda_step_block()).for_each([=](auto state) {
		std::size_t collected = 0;

		for(std::size_t i = 0; i < NUM_COPIES; i++) {
			auto shm_state = state.template with<noarr::cuda_stripe_index>((i + my_copy_idx) % NUM_COPIES);
			collected += shm_struct | noarr::get_at(shm_ptr, shm_state);
		}

		auto &bin = out_struct | noarr::get_at(out_ptr, state);
		atomicAdd(&bin, collected);
	});
}

void histo_cuda(void *in_ptr, std::size_t size, void *out_ptr) {
	auto in = noarr::scalar<value_t>() ^ noarr::sized_vector<'i'>(size);
	auto out = noarr::scalar<std::size_t>() ^ noarr::array<'v', NUM_VALUES>();

	auto in_blk = in ^ noarr::into_blocks_static<'i', 'C', 'y', 'z'>(BLOCK_SIZE) ^ noarr::into_blocks_static<'y', 'D', 'x', 'y'>(ELEMS_PER_THREAD);
	auto out_striped = out ^ noarr::cuda_striped<NUM_COPIES>();

	noarr::traverser(in_blk).order(noarr::reorder<'C', 'D'>()).for_each([=](auto cd){
		auto ct = noarr::cuda_traverser(in_blk).order(noarr::fix(cd)).template threads<'x', 'z'>();
#ifdef NOARR_CUDA_HISTO_DEBUG
		std::cerr
			<< (noarr::get_index<'C'>(cd) ? "border" : "body")
			<< " of "
			<< (noarr::get_index<'D'>(cd) ? "border" : "body")
			<< ": len<x> = gridDim = "  << (in_blk ^ noarr::fix(cd) | noarr::get_length<'x'>())
			<< ", len<y> = loopLen = "  << (in_blk ^ noarr::fix(cd) | noarr::get_length<'y'>())
			<< ", len<z> = blockDim = " << (in_blk ^ noarr::fix(cd) | noarr::get_length<'z'>())
			<< std::endl;
		std::cerr << (ct?"if(true)\t":"if(false)\t") << "kernel_histo<<<" << ct.grid_dim().x << ", " << ct.block_dim().x << ", " << (out_striped|noarr::get_size()) << ">>>(...);" <<  << std::endl;
#endif
		if(!ct) return;
		kernel_histo<<<ct.grid_dim(), ct.block_dim(),out_striped|noarr::get_size()>>>(ct.inner(), in_blk, out_striped, out, in_ptr, out_ptr);
		CUCH(hipGetLastError());
#ifdef NOARR_CUDA_HISTO_DEBUG
		CUCH(hipDeviceSynchronize());
#endif
	});

	CUCH(hipDeviceSynchronize());
}
