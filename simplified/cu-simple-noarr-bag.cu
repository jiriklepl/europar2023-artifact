#include "hip/hip_runtime.h"
#define CUDA
#include "../histo/histomain.hpp"

#include <noarr/structures_extended.hpp>
#include <noarr/structures/extra/traverser.hpp>
#include <noarr/structures/interop/bag.hpp>
#include <noarr/structures/interop/cuda_traverser.cuh>
#include <noarr/structures/interop/cuda_striped.cuh>
#include <noarr/structures/interop/cuda_step.cuh>

// PAPER: 4.2 First example
template<class InT, class In, class ShmS, class Out>
__global__ void histogram(InT in_trav, In in, ShmS shm_s, Out out) {
	extern __shared__ char shm_ptr[];
	auto shm_bag = make_bag(shm_s, shm_ptr);

	// PAPER: 4.2 Third example
	// A private copy will usually be shared by multiple threads (whenever NUM_COPIES < blockDim.x).
	// For some actions, we would like each memory location to be assigned to only one thread.
	// Let us split each copy further into "subsets", where each subset is owned by exactly one thread.
	// Note that `shm_bag` uses `threadIdx%NUM_COPIES` as the index of copy.
	// We can use the remaining bits, `threadIdx/NUM_COPIES`, as the index of subset within copy.
	std::size_t my_copy_idx = shm_s.current_stripe_index();
	auto subset = noarr::cuda_step(shm_s.current_stripe_cg());

	// Zero out shared memory. In this particular case, the access pattern happens
	// to be the same as with the `for(i = threadIdx; i < ...; i += blockDim)` idiom.
	noarr::traverser(shm_bag).order(subset).for_each([=](auto state) {
		shm_bag[state] = 0;
	});

	__syncthreads();

	// Count the elements into the histogram copies in shared memory.
	in_trav.for_each([=](auto state) {
		auto value = in[state];
		atomicAdd(&shm_bag[noarr::idx<'v'>(value)], 1);
	});

	__syncthreads();

	// PAPER: 4.2 Fourth example
	// Reduce the bins in shared memory into global memory.
	noarr::traverser(out)
		.order(noarr::cuda_step_block())
		.for_each([=](auto state) {
			std::size_t sum = 0;

			for(std::size_t i = 0; i < shm_s.num_stripes(); i++) {
				auto shm_state = state.template with<noarr::cuda_stripe_index>((i + my_copy_idx) % shm_s.num_stripes());
				sum += shm_bag[shm_state];
			}

			atomicAdd(&out[state], sum);
	});
}

void run_histogram(value_t *in_ptr, std::size_t size, std::size_t *out_ptr) {
	auto in_struct = noarr::scalar<value_t>() ^ noarr::sized_vector<'i'>(size);
	auto out_struct = noarr::scalar<std::size_t>() ^ noarr::array<'v', NUM_VALUES>();

	// PAPER: 4.1 Second example
	auto in_blk_struct = in_struct
		^ noarr::into_blocks<'i', 'B', 't'>(noarr::lit<BLOCK_SIZE>)
		^ noarr::into_blocks<'B', 'b', 'x'>(noarr::lit<ELEMS_PER_THREAD>);
	auto in = noarr::make_bag(in_blk_struct, in_ptr);
	auto out = noarr::make_bag(out_struct, out_ptr);

	// PAPER: 4.2 Second example
	auto ct = noarr::cuda_threads<'b', 't'>(noarr::traverser(in));
	auto shm_s = out_struct ^ noarr::cuda_striped<NUM_COPIES>();

	// Shorter version of
	//   histogram<<<ct.grid_dim(), ct.block_dim(), shm_s | noarr::get_size()>>>(ct.inner(), in, shm_s, out);
	ct.simple_run(histogram, shm_s | noarr::get_size(), in, shm_s, out);
	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());
}
